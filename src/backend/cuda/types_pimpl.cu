#include "types_pimpl.hpp"
#include <dvcxx/backend/cuda/exceptions.hpp>

namespace dvcxx  {
namespace cuda   {
namespace detail {


managed_event_pimpl::managed_event_pimpl() {
  CUDA_THROW( hipEventCreate( &this->event ) );
}

managed_event_pimpl::~managed_event_pimpl() noexcept {
  CUDA_ASSERT( hipEventDestroy( this->event ) );
}





managed_stream_pimpl::managed_stream_pimpl() {
  CUDA_THROW( hipStreamCreate( &this->stream ) );
}

managed_stream_pimpl::~managed_stream_pimpl() noexcept {
  CUDA_ASSERT( hipStreamDestroy( this->stream ) );
}





pointer_attributes_pimpl::pointer_attributes_pimpl( const void* p ) : ptr( p ) {

  auto ret = hipPointerGetAttributes( &attributes, ptr );
  if( ret != hipSuccess and ret != hipErrorInvalidValue )
    CUDA_THROW( ret )
  else if( ret == hipErrorInvalidValue )
    attributes.type = cudaMemoryTypeUnregistered;

}




} // namespace detail
} // namespace cuda
} // namespace dvcxx
